#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>



typedef double (*benchmark_func_t)(void);

typedef long   int_t;
typedef double real_t;



/**
 * Helper function to get current timestamp,
 * taken from STREAM Triad.
 */
double mysecond()
{
    struct timeval tp;
    struct timezone tzp;
    gettimeofday(&tp, &tzp);

    return (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6;
}



/**
 * Integer matrix multiplication benchmark.
 *
 * See also:
 *
 *   https://www.math.utah.edu/~mayer/linux/bmark.html
 */
#define MATMUL_CPU_ARRAY_SIZE (1<<9)
#define MATMUL_CPU_NTIMES 4

#define ELEM(M, n, i, j) ((M)[(i) * (n) + (j)])

double benchmark_cpu_iops()
{
    int n = MATMUL_CPU_ARRAY_SIZE;
    int_t * A = (int_t *)malloc(n * n * sizeof(int_t));
    int_t * B = (int_t *)malloc(n * n * sizeof(int_t));
    int_t * C = (int_t *)malloc(n * n * sizeof(int_t));

    for ( int i = 0; i < MATMUL_CPU_ARRAY_SIZE; i++ )
    {
        for ( int j = 0; j < MATMUL_CPU_ARRAY_SIZE; j++ )
        {
            ELEM(A, n, i, j) = 1;
            ELEM(B, n, i, j) = 2;
            ELEM(C, n, i, j) = 0;
        }
    }

    double iops = 2.0 * n * n * n;
    double min_time = INFINITY;

    for ( int l = 0; l < MATMUL_CPU_NTIMES; l++ )
    {
        double t = mysecond();

        for ( int i = 0; i < MATMUL_CPU_ARRAY_SIZE; i++ )
        {
            for ( int j = 0; j < MATMUL_CPU_ARRAY_SIZE; j++ )
            {
                ELEM(C, n, i, j) = 0;

                for ( int k = 0; k < MATMUL_CPU_ARRAY_SIZE; k++ )
                {
                    ELEM(C, n, i, j) += ELEM(A, n, i, j) * ELEM(B, n, i, j);
                }
            }
        }

        t = mysecond() - t;

#ifdef VERBOSE
        printf("%f\n", t);
#endif

        if ( l > 0 && t < min_time )
        {
            min_time = t;
        }
    }

    free(A);
    free(B);
    free(C);

    return iops / min_time / 1e9;
}



/**
 * Floating-point matrix multiplication benchmark based on HPL.
 *
 *   http://www.netlib.org/benchmark/hpl/
 */
double benchmark_cpu_flops()
{
    int n = MATMUL_CPU_ARRAY_SIZE;
    real_t * A = (real_t *)malloc(n * n * sizeof(real_t));
    real_t * B = (real_t *)malloc(n * n * sizeof(real_t));
    real_t * C = (real_t *)malloc(n * n * sizeof(real_t));

    for ( int i = 0; i < MATMUL_CPU_ARRAY_SIZE; i++ )
    {
        for ( int j = 0; j < MATMUL_CPU_ARRAY_SIZE; j++ )
        {
            ELEM(A, n, i, j) = 1.0;
            ELEM(B, n, i, j) = 2.0;
            ELEM(C, n, i, j) = 0.0;
        }
    }

    double flops = 2.0 * n * n * n;
    double min_time = INFINITY;

    for ( int l = 0; l < MATMUL_CPU_NTIMES; l++ )
    {
        double t = mysecond();

        for ( int i = 0; i < MATMUL_CPU_ARRAY_SIZE; i++ )
        {
            for ( int j = 0; j < MATMUL_CPU_ARRAY_SIZE; j++ )
            {
                ELEM(C, n, i, j) = 0.0;

                for ( int k = 0; k < MATMUL_CPU_ARRAY_SIZE; k++ )
                {
                    ELEM(C, n, i, j) += ELEM(A, n, i, j) * ELEM(B, n, i, j);
                }
            }
        }

        t = mysecond() - t;

#ifdef VERBOSE
        printf("%f\n", t);
#endif

        if ( l > 0 && t < min_time )
        {
            min_time = t;
        }
    }

    free(A);
    free(B);
    free(C);

    return flops / min_time / 1e9;
}



/**
 * Vector arithmetic benchmark based on STREAM Triad.
 *
 *   http://www.cs.virginia.edu/stream/
 */
#define STREAM_ARRAY_SIZE 10000000
#define STREAM_NTIMES 4

double benchmark_cpu_mem_bw()
{
    real_t * a = (real_t *)malloc(STREAM_ARRAY_SIZE * sizeof(real_t));
    real_t * b = (real_t *)malloc(STREAM_ARRAY_SIZE * sizeof(real_t));
    real_t * c = (real_t *)malloc(STREAM_ARRAY_SIZE * sizeof(real_t));
    real_t scalar = 3.0f;

    for ( int j = 0; j < STREAM_ARRAY_SIZE; j++ )
    {
        a[j] = 1.0;
        b[j] = 2.0;
        c[j] = 0.0;
    }

    double bytes = 3.0 * sizeof(real_t) * STREAM_ARRAY_SIZE;
    double min_time = INFINITY;

    for ( int k = 0; k < STREAM_NTIMES; k++ )
    {
        double t = mysecond();

        for ( int j = 0; j < STREAM_ARRAY_SIZE; j++ )
        {
            a[j] = b[j] + scalar * c[j];
        }

        t = mysecond() - t;

#ifdef VERBOSE
        printf("%f\n", t);
#endif

        if ( k > 0 && t < min_time )
        {
            min_time = t;
        }
    }

    free(a);
    free(b);
    free(c);

    return bytes / min_time / (1 << 30);
}



/**
 * Read a file from disk.
 */
#define READ_FILE_SIZE (1<<30)
#define READ_NTIMES 4

double benchmark_disk_read()
{
    const char * filename = "tmp";
    FILE * file;
    char * data = (char *)malloc(READ_FILE_SIZE * sizeof(char));

    for ( int i = 0; i < READ_FILE_SIZE; i++ )
    {
        data[i] = rand();
    }

    file = fopen(filename, "w");
    fwrite(data, sizeof(char), READ_FILE_SIZE, file);
    fclose(file);

    double min_time = INFINITY;

    for ( int k = 0; k < READ_NTIMES; k++ )
    {
        double t = mysecond();

        file = fopen(filename, "r");
        fread(data, sizeof(char), READ_FILE_SIZE, file);
        fclose(file);

        t = mysecond() - t;

#ifdef VERBOSE
        printf("%f\n", t);
#endif

        if ( k > 0 && t < min_time )
        {
            min_time = t;
        }
    }

    free(data);
    remove(filename);

    return READ_FILE_SIZE / min_time / 1e9;
}



/**
 * Write a file to disk.
 */
#define WRITE_FILE_SIZE (1<<30)
#define WRITE_NTIMES 4

double benchmark_disk_write()
{
    const char * filename = "tmp";
    FILE * file;
    char * data = (char *)malloc(WRITE_FILE_SIZE * sizeof(char));

    for ( int i = 0; i < WRITE_FILE_SIZE; i++ )
    {
        data[i] = rand();
    }

    double min_time = INFINITY;

    for ( int k = 0; k < WRITE_NTIMES; k++ )
    {
        double t = mysecond();

        file = fopen(filename, "w");
        fwrite(data, sizeof(char), WRITE_FILE_SIZE, file);
        fclose(file);

        t = mysecond() - t;

#ifdef VERBOSE
        printf("%f\n", t);
#endif

        if ( k > 0 && t < min_time )
        {
            min_time = t;
        }
    }

    free(data);
    remove(filename);

    return WRITE_FILE_SIZE / min_time / 1e9;
}



/**
 * GPU kernel for gpu_flops benchmark.
 */
#define TILE_DIM 16

__global__
void benchmark_gpu_flops_kernel(int n, real_t * A, real_t * B, real_t * C)
{
    // blockDim.x = TILE_DIM
    // blockDim.y = TILE_DIM
    __shared__ real_t tile_A[TILE_DIM][TILE_DIM];
    __shared__ real_t tile_B[TILE_DIM][TILE_DIM];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int offset_x = blockIdx.x * blockDim.x + tx;
    int offset_y = blockIdx.y * blockDim.y + ty;
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for ( int i = offset_y; i < n; i += stride_y ) {
        for ( int j = offset_x; j < n; j += stride_x ) {
            real_t C_ij = 0;

            // iterate through each tile pair in A, B
            for ( int offset_t = 0; offset_t < n; offset_t += TILE_DIM ) {
                // load tiles into shared memory
                tile_A[ty][tx] = A[i * n + (offset_t + tx)];
                tile_B[ty][tx] = B[(offset_t + ty) * n + j];

                __syncthreads();

                // update sum of products
                for ( int k = 0; k < TILE_DIM; k++ ) {
                    C_ij += tile_A[ty][k] * tile_B[k][tx];
                }

                __syncthreads();
            }

            // save output value
            C[i * n + j] = C_ij;
        }
    }
}



/**
 * Floating-point matrix multiplication benchmark based on HPL.
 *
 *   http://www.netlib.org/benchmark/hpl/
 */
#define MATMUL_GPU_ARRAY_SIZE (1<<12)
#define MATMUL_GPU_NTIMES 4

double benchmark_gpu_flops()
{
    int n_devices;
    hipGetDeviceCount(&n_devices);

    if ( n_devices == 0 ) {
        return 0.0;
    }

    int n = MATMUL_GPU_ARRAY_SIZE;
    real_t * A = (real_t *)malloc(n * n * sizeof(real_t));
    real_t * B = (real_t *)malloc(n * n * sizeof(real_t));
    real_t * C = (real_t *)malloc(n * n * sizeof(real_t));

    real_t * d_A;
    real_t * d_B;
    real_t * d_C;
    hipMalloc(&d_A, n * n * sizeof(real_t));
    hipMalloc(&d_B, n * n * sizeof(real_t));
    hipMalloc(&d_C, n * n * sizeof(real_t));

    for ( int i = 0; i < MATMUL_GPU_ARRAY_SIZE; i++ )
    {
        for ( int j = 0; j < MATMUL_GPU_ARRAY_SIZE; j++ )
        {
            ELEM(A, n, i, j) = 1.0;
            ELEM(B, n, i, j) = 2.0;
            ELEM(C, n, i, j) = 0.0;
        }
    }

    hipMemcpy(d_A, A, n * n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * n * sizeof(real_t), hipMemcpyHostToDevice);

    double flops = 2.0 * n * n * n;
    double min_time = INFINITY;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int nSMs;
    hipDeviceGetAttribute(&nSMs, hipDeviceAttributeMultiprocessorCount, 0);

    for ( int l = 0; l < MATMUL_GPU_NTIMES; l++ )
    {
        hipEventRecord(start);

        dim3 block(TILE_DIM, TILE_DIM);
        dim3 grid(2 * nSMs, 4);
        benchmark_gpu_flops_kernel<<<grid, block>>>(n, d_A, d_B, d_C);

        hipEventRecord(stop);

        hipMemcpy(C, d_C, n * n * sizeof(real_t), hipMemcpyDeviceToHost);

        float t;
        hipEventElapsedTime(&t, start, stop);

#ifdef VERBOSE
        printf("%f\n", t / 1000);

        // double max_error = 0.0;
        // for ( int i = 0; i < n * n; i++ ) {
        //     max_error = max(max_error, abs(C[i] - 2.0 * n));
        // }

        // printf("max_error = %f\n", max_error);
#endif

        if ( l > 0 && t < min_time )
        {
            min_time = t;
        }
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    return flops / min_time / 1e6;
}



/**
 * GPU kernel for gpu_mem_bw benchmark.
 */
__global__
void benchmark_gpu_mem_bw_kernel(int n, real_t * a, real_t * b, real_t * c, real_t scalar)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for ( int i = offset; i < n; i += stride ) {
        a[i] = b[i] + scalar * c[i];
    }
}



/**
 * Vector arithmetic benchmark based on STREAM Triad.
 *
 *   http://www.cs.virginia.edu/stream/
 */
double benchmark_gpu_mem_bw()
{
    int n_devices;
    hipGetDeviceCount(&n_devices);

    if ( n_devices == 0 ) {
        return 0.0;
    }

    int n = STREAM_ARRAY_SIZE;
    real_t * a = (real_t *)malloc(n * sizeof(real_t));
    real_t * b = (real_t *)malloc(n * sizeof(real_t));
    real_t * c = (real_t *)malloc(n * sizeof(real_t));
    real_t scalar = 3.0f;

    real_t * d_a;
    real_t * d_b;
    real_t * d_c;
    hipMalloc(&d_a, n * sizeof(real_t));
    hipMalloc(&d_b, n * sizeof(real_t));
    hipMalloc(&d_c, n * sizeof(real_t));

    for ( int j = 0; j < n; j++ )
    {
        a[j] = 1.0;
        b[j] = 2.0;
        c[j] = 0.0;
    }

    hipMemcpy(d_a, a, n * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(real_t), hipMemcpyHostToDevice);

    double bytes = 3.0 * sizeof(real_t) * n;
    double min_time = INFINITY;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int nSMs;
    hipDeviceGetAttribute(&nSMs, hipDeviceAttributeMultiprocessorCount, 0);

    for ( int k = 0; k < STREAM_NTIMES; k++ )
    {
        hipEventRecord(start);

        dim3 block(256);
        dim3 grid(8 * nSMs);
        benchmark_gpu_mem_bw_kernel<<<grid, block>>>(n, d_a, d_b, d_c, scalar);

        hipEventRecord(stop);

        hipMemcpy(a, d_a, n * sizeof(real_t), hipMemcpyDeviceToHost);

        float t;
        hipEventElapsedTime(&t, start, stop);

#ifdef VERBOSE
        printf("%f\n", t / 1000);

        // double max_error = 0.0;
        // for ( int i = 0; i < n; i++ ) {
        //     max_error = max(max_error, abs(a[i] - 2.0));
        // }

        // printf("max_error = %f\n", max_error);
#endif

        if ( k > 0 && t < min_time )
        {
            min_time = t;
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return bytes / min_time / 1e6;
}



typedef struct {
    const char * name;
    const char * format;
} format_t;



typedef struct {
    const char * name;
    benchmark_func_t func;
} benchmark_t;



int main(int argc, char **argv)
{
    // parse command-line arguments
    if ( argc != 2 )
    {
        fprintf(stderr, "usage: ./minibench <output-format>\n");
        exit(-1);
    }

    char * fmt_name = argv[1];

    // define output formats
    format_t formats[] = {
        { "csv",   "%s\t%0.6f" },
        { "trace", "#TRACE %s=%0.6f" }
    };
    int n_formats = sizeof(formats) / sizeof(format_t);

    // select output format
    format_t * fmt = NULL;

    for ( int i = 0; i < n_formats; i++ )
    {
        if ( strcmp(formats[i].name, fmt_name) == 0 )
        {
            fmt = &formats[i];
        }
    }

    if ( fmt == NULL )
    {
        fprintf(stderr, "error: invalid output format %s\n", fmt_name);
        exit(-1);
    }

    // define benchmarks
    benchmark_t benchmarks[] = {
        { "cpu_iops",   benchmark_cpu_iops },   // GIOP/s
        { "cpu_flops",  benchmark_cpu_flops },  // GFLOP/s
        { "cpu_mem_bw", benchmark_cpu_mem_bw }, // GiB/s
        { "disk_read",  benchmark_disk_read },  // GB/s
        { "disk_write", benchmark_disk_write }, // GB/s
        { "gpu_flops",  benchmark_gpu_flops },  // GFLOPS/s
        { "gpu_mem_bw", benchmark_gpu_mem_bw }  // GB/s
    };
    int n_benchmarks = sizeof(benchmarks) / sizeof(benchmark_t);

    // run benchmarks
    for ( int i = 0; i < n_benchmarks; i++ )
    {
        benchmark_t *b = &benchmarks[i];

        printf(fmt->format, b->name, b->func());
        printf("\n");
    }

    return 0;
}